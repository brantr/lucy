#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdlib.h>
#include <stdio.h>
#include "cuda_convolve.h"
#include "global.h"

#define TPB 128

/*
hipfftHandle plan;
hipfftComplex *dev_x;
hipfftComplex *dev_y;
hipfftComplex *dev_z;
*/

int nrank;
int *n_rank;

void destroy_cuda_convolve(void)
{
  /* Destroy the CUFFT plan */
  //hipfftDestroy(plan);

}
void initialize_cuda_convolve_2d(int nx, int ny)
{
  /* set rank */
  nrank = 2;

  n_rank = (int *) malloc(nrank*sizeof(int));

  n_rank[0] = nx;
  n_rank[1] = ny;
  
}

void cuda_fft_test(void)
{
  int BATCH = 1;
  int NX=512;
  int NY=512;
  int NZ=128;
  hipfftHandle plan;
  hipfftComplex *data ;
  //int n[3] = {NX, NY, NZ};
  int n[2] = {NX, NY};
  hipMalloc((void**)&data, sizeof(hipfftComplex)*NX*NY*NZ*BATCH);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    return;
  }
  // Create a 3D FFT plan. 
  /*if ( hipfftPlanMany(&plan , 3, n , NULL, 1, NX*NY*NZ, // ∗inembed, istride , idist  
					 NULL, 1, NX*NY*NZ, // *onembed, ostride, odist
					 HIPFFT_C2C , BATCH ) != HIPFFT_SUCCESS ) { */
  if ( hipfftPlanMany(&plan , 2, n , NULL, 1, NX*NY, // ∗inembed, istride , idist  
					 NULL, 1, NX*NY, // *onembed, ostride, odist
					 HIPFFT_C2C , BATCH ) != HIPFFT_SUCCESS ) { 
    fprintf(stderr, "CUFFT error: Plan creation failed");
    return;
  }

  // Use the CUFFT plan to transform the signal in place . 
  if (hipfftExecC2C(plan , data , data , HIPFFT_FORWARD) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
    return ;
  }

  // Inverse transform the signal in place. 
  if (hipfftExecC2C(plan , data , data , HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
    return ;
  }
  // Note : (1) Divide by number of elements in data set to get back original data (2) Identical pointers to input and output arrays implies in−place transformation
  if (hipDeviceSynchronize() != hipSuccess)
  {
    fprintf(stderr, "Cuda error\n");
    return;
  }

  /* Destroy the CUFFT plan . */
  hipfftDestroy ( plan ); 
  hipFree ( data ) ;
}

Real **cuda_convolve_2d(Real **x, Real **y, int nx, int ny)
{
  Real **z;

  hipfftHandle plan;
  //hipfftHandle iplan;
  hipfftComplex *dev_x;
  hipfftComplex *dev_y;
  hipfftComplex *dev_z;
  hipfftComplex *host_x;
  hipfftComplex *host_y;
  hipfftComplex *host_z;

  dim3 dimGrid(nx*ny/TPB,1,1);
  dim3 dimBlock(TPB,1,1);


  z      = allocate_2d_array(nx,ny);

  host_x = (hipfftComplex *) malloc( nx*ny *sizeof(hipfftComplex) );
  host_y = (hipfftComplex *) malloc( nx*ny *sizeof(hipfftComplex) );
  host_z = (hipfftComplex *) malloc( nx*ny *sizeof(hipfftComplex) );

  printf("CUDA Convolve...\n");

  hipMalloc((void**)&dev_x, sizeof(hipfftComplex)*nx*ny);
  if(hipGetLastError() != hipSuccess)
  {
    fprintf(stderr, "Cuda Error: Failed to allocate dev_x\n");
  }

  hipMalloc((void**)&dev_y, sizeof(hipfftComplex)*nx*ny);
  if(hipGetLastError() != hipSuccess)
  {
    fprintf(stderr, "Cuda Error: Failed to allocate dev_y\n");
  }

  hipMalloc((void**)&dev_z, sizeof(hipfftComplex)*nx*ny);
  if(hipGetLastError() != hipSuccess)
  {
    fprintf(stderr, "Cuda Error: Failed to allocate dev_z\n");
  }


  if( hipfftPlanMany(&plan, nrank, n_rank, NULL, 1, nx*ny, NULL, 1, nx*ny, HIPFFT_C2C, 1) != HIPFFT_SUCCESS)
  {
    fprintf(stderr, "CUFFT Error: Unable to create plans\n");
    return NULL;
  }

  for(int i=0;i<nx;i++)
    for(int j=0;j<ny;j++)
    {
      host_x[nx*j + i].x = x[i][j];
      host_y[nx*j + i].x = y[i][j];

      if(isnan(x[i][j]))
      {
	printf("XNAN! i %d j %d\n",i,j);
	fflush(stdout);
	exit(0);
      }
      if(isnan(y[i][j]))
	printf("YNAN! i %d j %d\n",i,j);
    }


  hipMemcpy(dev_x, host_x, nx*ny*sizeof(hipfftComplex), hipMemcpyHostToDevice);
  hipMemcpy(dev_y, host_y, nx*ny*sizeof(hipfftComplex), hipMemcpyHostToDevice);

  // Transform in place 
  if( hipfftExecC2C(plan, dev_x, dev_x, HIPFFT_FORWARD) != HIPFFT_SUCCESS )
  {
    fprintf(stderr, "CUFFT Error: ExecC2C Forward on dev_x failed %d\n",hipGetLastError());
    return NULL;
  }

  // Transform in place 
  if( hipfftExecC2C(plan, dev_y, dev_y, HIPFFT_FORWARD) != HIPFFT_SUCCESS )
  {
    fprintf(stderr, "CUFFT Error: ExecC2C Forward on dev_y failed %d\n",hipGetLastError());
    return NULL;
  }

  printf("Before convolution nx = %d ny = %d\n",nx,ny);
  // do convolution 
  convolve_2d_CUDA<<<dimGrid,dimBlock>>>(dev_x, dev_y, dev_z, nx, ny);

  // Transform in place 

  if( hipfftExecC2C(plan, dev_z, dev_z, HIPFFT_BACKWARD) != HIPFFT_SUCCESS )
  {
    fprintf(stderr, "CUFFT Error: ExecC2C Inverse on dev_z failed %d\n",hipGetLastError());
    return NULL;
  }


  hipDeviceSynchronize();

  // To CPU 
  hipMemcpy(host_z, dev_z, nx*ny*sizeof(hipfftComplex), hipMemcpyDeviceToHost);

  for(int i=0;i<nx;i++)
    for(int j=0;j<ny;j++)
    {
      if(isnan(x[i][j]))
      {
	printf("XNAN! i %d j %d\n",i,j);
	fflush(stdout);
	exit(0);
      }
      if(x[i][j]<0)
      {
	printf("XNEG! i %d j %d\n",i,j);
	fflush(stdout);
	exit(0);
      }
    }
  for(int i=0;i<nx;i++)
    for(int j=0;j<ny;j++)
    {
      if(y[i][j]<0)
      {
	printf("YNEG! i %d j %d\n",i,j);
	fflush(stdout);
	exit(0);
      }
      if(isnan(y[i][j]))
      {
	printf("YNAN! i %d j %d\n",i,j);
	fflush(stdout);
	exit(0);
      }
    }
    
  for(int i=0;i<nx;i++)
    for(int j=0;j<ny;j++)
    {
      z[i][j] = host_z[nx*j + i].x;
      if(isnan(z[i][j]))
      {
	printf("ZNAN! i %d j %d\n",i,j);
	fflush(stdout);
//	exit(0);
      }
    }

  hipfftDestroy(plan);

  hipFree(dev_x);
  hipFree(dev_y);
  hipFree(dev_z);

  free(host_x);
  free(host_y);
  free(host_z);

  // To GPU 
  // return result 
  return z;
  
}

__global__ void convolve_2d_CUDA(hipfftComplex *dev_x, hipfftComplex *dev_y, hipfftComplex *dev_z, int nx, int ny)
{
  Real scale = (Real) (1./( ((Real) nx)*((Real) ny) ));
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if(idx<nx*ny)
  {
    dev_z[idx].x = (dev_x[idx].x*dev_y[idx].x - dev_x[idx].y*dev_y[idx].y)*scale;
    dev_z[idx].y = (dev_x[idx].x*dev_y[idx].y + dev_x[idx].y*dev_y[idx].x)*scale;
    //dev_z[idx].x = dev_x[idx].x;
    //dev_z[idx].y = dev_x[idx].y;
    //dev_z[idx].x = 1;
    //dev_z[idx].y = 0;
  }
}
